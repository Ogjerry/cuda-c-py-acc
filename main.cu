/*
 * execution program for the whole project. Have no idea what project to carry on for now.
 * Will figure out lately. This document is a playground for cuda C/C++ programming.
 * Possible future extensions include improving computing speed of large statistical learning
 * models or computing matrix operations, which is the related field of my major study.

*/

#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        printf("Thread %d: BlockIdx %d, ThreadIdx %d\n", i, blockIdx.x, threadIdx.x);
        printf("Thread %d: %f + %f = %f\n", i, a[i], b[i], a[i] + b[i]);
        c[i] = a[i] + b[i];
    }
}


int main() {
    int n = 256; // Size of the vectors
    int size = n * sizeof(float);

    // Allocate host memory
    float *h_a = (float*)malloc(size);
    float *h_b = (float*)malloc(size);
    float *h_c = (float*)malloc(size);

    // Initialize input vectors on the host
    for (int i = 0; i < n; ++i) {
        h_a[i] = i;
        h_b[i] = 2 * i;
    }

    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Copy input vectors from host to GPU
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    
    // Launch the vectorAdd kernel on the GPU
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    vectorAdd<<<numBlocks, blockSize>>>(d_a, d_b, d_c, n);

    // Print some values from device arrays
    for (int i = 0; i < n; ++i) {
        float a_val, b_val, c_val;
        hipMemcpy(&a_val, d_a + i, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&b_val, d_b + i, sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(&c_val, d_c + i, sizeof(float), hipMemcpyDeviceToHost);
        printf("Device Array Values %d: %f, %f, %f\n", i, a_val, b_val, c_val);
    }

    // Copy the result from GPU to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    // Print some values from device arrays
    for (int i = 0; i < n; ++i) {
        float c_val;
        hipMemcpy(&c_val, d_c + i, sizeof(float), hipMemcpyDeviceToHost);
        // printf("Device Array Values %d: %f\n", i, c_val);
        
        // Print intermediate values
        printf("Intermediate Result %d: %f\n", i, h_c[i]);
    }

    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
