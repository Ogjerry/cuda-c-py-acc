#include <stdio.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(call)                                                          \
{                                                                                 \
    const hipError_t error = call;                                               \
    if (error != hipSuccess)                                                     \
    {                                                                             \
        printf("Error: %s:%d, ", __FILE__, __LINE__);                             \
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));       \
        exit(1);                                                                  \
    }                                                                             \
}

__global__ void simpleKernel() {
    printf("Simple kernel executed on device.\n");
}

int main(int argc, char const *argv[])
{
    printf("Setting up device...\n");
    int dev = 0;
    hipDeviceProp_t device_prop;
    CUDA_CHECK(hipGetDeviceProperties(&device_prop, dev));
    printf("Starting reduction at device %d: %s\n", dev, device_prop.name);
    CUDA_CHECK(hipSetDevice(dev));

    printf("Running a simple kernel for debugging...\n");
    simpleKernel<<<1, 1>>>();
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());

    printf("Simple kernel completed.\n");

    CUDA_CHECK(hipDeviceReset());

    return 0;
}
